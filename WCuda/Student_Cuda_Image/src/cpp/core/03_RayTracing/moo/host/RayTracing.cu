#include "hip/hip_runtime.h"
#include <iostream>
#include "Device.h"
#include "RayTracing.h"
#include <assert.h>
#include "Sphere.h"
#include "SphereCreator.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void raytracing_GM(uchar4* ptrDevPixels, uint w, uint h, float t, int nbSphere, Sphere* ptrDevTabSphere);
extern __global__ void raytracing_CM(uchar4* ptrDevPixels, uint w, uint h, float t);
extern __global__ void raytracing_SM(uchar4* ptrDevPixels, uint w, uint h, float t, int nbSphere, Sphere* ptrDevTabSphere);

extern __host__ void uploadGPU(Sphere* SphereTab);
/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

RayTracing::RayTracing(const Grid& grid, uint w, uint h, float dt, int nbSphere) :
	Animable_I<uchar4>(grid, w, h, "RayTracing_Cuda_RGBA_uchar4_Jeanneret_cyril"), i(0)
    {

    // Inputs
    this->dt = dt;

    // Tools
    this->t = 0; // protected dans Animable

    // Tools
    this->nbSphere = nbSphere;
    this->sizeOctetSpheres = sizeof(Sphere) * nbSphere;

    SphereCreator sphereCreator = SphereCreator(nbSphere, w, h);
    Sphere* ptrTabSphere = sphereCreator.getTabSphere();

    //GM transfert
    toGM(ptrTabSphere);

    //CM transfert
    toCM(ptrTabSphere);
    }

RayTracing::~RayTracing()
    {
    // rien
    Device::free(ptrDevTabSphere);
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 * Call periodicly by the API
 *
 * Note : domaineMath pas use car pas zoomable
 */

void RayTracing::toCM(Sphere* ptrTab)
    {
    uploadGPU(ptrTab);
    }

void RayTracing::toGM(Sphere* ptrTab)
    {
    Device::malloc(&ptrDevTabSphere, sizeOctetSpheres);
    Device::memclear(ptrDevTabSphere, sizeOctetSpheres);
    Device::memcpyHToD(ptrDevTabSphere, ptrTab, sizeOctetSpheres);

    }

void RayTracing::process(uchar4* ptrDevPixels, uint w, uint h, const DomaineMath& domaineMath)
    {
    Device::lastCudaError("RayTracing rgba uchar4 (before kernel)"); // facultatif, for debug only, remove for release

    // TODO lancer le kernel avec <<<dg,db>>>
    // le kernel est importer ci-dessus (ligne 19)

    if (i % 3 == 0)
	{
    raytracing_GM<<<dg,db>>>(ptrDevPixels, w, h, t, this->nbSphere, this->ptrDevTabSphere);
    }
else if (i % 3 == 1)
    {
raytracing_CM<<<dg,db>>>(ptrDevPixels, w, h, t);
}
else if (i % 3 == 2)
{
raytracing_SM<<<dg,db, this->sizeOctetSpheres>>>(ptrDevPixels, w, h, t, this->nbSphere, this->ptrDevTabSphere);
}
this->i++;

Device::lastCudaError("RayTracing rgba uchar4 (after kernel)"); // facultatif, for debug only, remove for release
}

/**
 * Override
 * Call periodicly by the API
 */
void RayTracing::animationStep()
{
t += dt;
}

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

