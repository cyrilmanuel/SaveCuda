#include "hip/hip_runtime.h"
#include "Indice1D.h"
#include "cudaTools.h"

__device__ void reductionIntraThread(int* tabSM, int* tabGM, int dataSizeSM);
__device__ void reductionInterBlock(int* tabSM, int* tabGM, int dataMax);
__device__ void initSM(int* tabSM, int max);
__device__ void resetGM(int* tabGM, int max);

__global__ void histoPerso(int* ptrDevData, int max, int size);

__global__ void histoPerso(int* ptrDevData, int max, int size)
    {
    extern __shared__ int tabSM[];

    initSM(tabSM, max);
    __syncthreads();
    reductionIntraThread(tabSM, ptrDevData, size);
    __syncthreads();

    resetGM(ptrDevData, max);
    __syncthreads();
    reductionInterBlock(tabSM, ptrDevData, max);
    }

__device__ void initSM(int* tabSM, int max)
    {
    const int TID_LOCAL = Indice1D::tidLocal();
    const int NB_THREAD_LOCAL = Indice1D::nbThreadBlock();

    int s = TID_LOCAL;

    while (s < max)
	{
	tabSM[s] = 0;
	s += NB_THREAD_LOCAL;
	}
    }

__device__ void reductionIntraThread(int* tabSM, int* tabGM, int dataSize)
    {
    const int TID = Indice1D::tid();
    const int NB_THREAD = Indice1D::nbThread();

    int s = TID;

    while (s < dataSize)
	{
	atomicAdd(&tabSM[tabGM[s]], 1);
	s += NB_THREAD;
	}
    }

__device__ void resetGM(int* tabGM, int max)
    {
    const int TID = Indice1D::tid();
    const int NB_THREAD = Indice1D::nbThread();

    int s = TID;

    while (s < max)
	{
	tabGM[s] = 0;
	s += NB_THREAD;
	}
    }

__device__ void reductionInterBlock(int* tabSM, int* tabGM, int dataMax)
    {
    const int TID = Indice1D::tid();
    const int NB_THREAD = Indice1D::nbThread();
    const int TID_LOCAL = Indice1D::tidLocal();
    const int NB_THREAD_LOCAL = Indice1D::nbThreadBlock();

    int s = TID_LOCAL;

    while (s < dataMax)
	{
	atomicAdd(&tabGM[s], tabSM[s]);
	s += NB_THREAD_LOCAL;
	}
    }
