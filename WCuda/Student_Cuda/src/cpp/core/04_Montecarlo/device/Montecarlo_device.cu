#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "Indice1D.h"
#include "cudaTools.h"

#include "reductionADD.h"
#include <stdio.h>

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void Montecarlo(float* ptrGMResultat, int nbSlice, float* ptrGenerator);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

static __device__ void reductionIntraThread(float* tabSM, int nbSlice, float* ptrGenerator);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/**
 * output : void required !!
 */
__global__ void Montecarlo(float* ptrGMResultat, int nbSlice, float* ptrGenerator)
    {
    extern __shared__ float tabSM[];
    reductionIntraThread(tabSM, nbSlice);
    __syncthreads();
    reductionADD<float>(tabSM, ptrGMResultat);
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__device__ float work(float x, float y);
    {
    // regarde si la fleche est en dessous ou en dessus
    return 1;
    }

__device__ void reductionIntraThread(float* tabSM, int nbSlice, float* ptrGenerator)
    {

    const int TID = Indice1D::tid();

    // Global Memory -> Register (optimization)
    hiprandState localGenerator = ptrGenerator[TID];

    float xAlea;
    float yAlea;
    for (long i = 1; i <= n; i++)
    {
    xAlea = hiprand_uniform(&localGenerator);
    yAlea = hiprand_uniform(&localGenerator);

    // la fonction work va pouvoir determiner si le thread (plus précisement la flechette)
    // se trouve en dessus ou en dessous dans le caré
    work(xAlea,yAlea);

    }

}
/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

